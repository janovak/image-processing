
#include <hip/hip_runtime.h>
#include <stdint.h>

#define RADIUS 25

__device__ unsigned int GetPixelWithPadding(unsigned int x, unsigned int y, unsigned int width, unsigned int padding)
{
    return (x + padding + (y + padding) * (width + 2 * padding)) * gridDim.z + blockIdx.z;
}

__device__ unsigned int GetPixel(unsigned int x, unsigned int y, unsigned int width)
{
    return GetPixelWithPadding(x, y, width, 0);
}

extern "C" __global__ void BoxBlur(uint8_t *in_array, uint8_t *out_array, unsigned int width, unsigned int height)
{
    unsigned int sum = 0;
    unsigned int denominator = 0;
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
    {
        return;
    }
    const unsigned int index = GetPixel(x, y, width);
#pragma unroll
    for (int dx = -RADIUS; dx <= RADIUS; ++dx)
    {
#pragma unroll
        for (int dy = -RADIUS; dy <= RADIUS; ++dy)
        {
            const int neighborX = x + dx;
            const int neighborY = y + dy;
            const int neighborIndex = GetPixelWithPadding(neighborX, neighborY, width, RADIUS);
            sum += in_array[neighborIndex];
            ++denominator;
        }
    }
    out_array[index] = sum / denominator;
}